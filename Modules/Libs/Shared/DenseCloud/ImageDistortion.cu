#include "hip/hip_runtime.h"
#include "ImageDistortion.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}
void ImageDistortion()
{
    float A[2] = { 0,1 };
    float B[2] = { 1,2 };
    float C[2] = {3,4 };
    VecAdd << <1, 2 >> > (A, B, C);
}
